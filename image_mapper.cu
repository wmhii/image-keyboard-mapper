
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdint.h>

#define ALPHA 0xFF000000
#define THREADS 512 
#define BLOCKS 256 

__global__ void imageMapKernel(uint8_t*out_image,
						  const uint8_t *colors_image,
						  const uint32_t *template_image,
						  const uint32_t *colors_list, 
						  unsigned int image_size,
						  unsigned int colors_size)
{
	uint32_t color = 0;
	float average_r;
	float average_g;
	float average_b;
	uint32_t compare_color = 0;
	
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < colors_size; i += blockDim.x * gridDim.x) {
		unsigned int x = 0;
		compare_color = colors_list[i];

		// Search for the first color
		for (x = 0; x < image_size; x++) {
			if (template_image[x] == compare_color) {
				average_r = (float)colors_image[x*4+0]; // future improvement, read from memory once and do the binary arithmetic
				average_g = (float)colors_image[x*4+1];
				average_b = (float)colors_image[x*4+2];
				break;
			}
		}
		// Average in the rest of the colors
		for (; x < image_size; x++) {
			if (template_image[x] == compare_color) {
				average_r = (average_r + (float)colors_image[x*4+0])/2.0f;
				average_g = (average_g + (float)colors_image[x*4+1])/2.0f;
				average_b = (average_b + (float)colors_image[x*4+2])/2.0f;
			}
		}

		// Write out the final averaged color
		for (x = 0; x < image_size; x++) {
			if (template_image[x] == compare_color) {
				out_image[x*4+0] = (uint8_t)average_r;
				out_image[x*4+1] = (uint8_t)average_g;
				out_image[x*4+2] = (uint8_t)average_b;
				out_image[x*4+3] = 255;
			}
		}
	}
}

extern "C" {
	__declspec(dllexport)

	hipError_t parallelImageTemplateMap(uint8_t* out_image,
		                                 uint8_t* colors_image,
		                                 uint8_t* template_image,
		                                 uint8_t* colors_list,
		                                 unsigned int image_size, 
		                                 unsigned int colors_size)
	{
		uint8_t *dev_out = 0;
		uint8_t *dev_colors = 0;
		uint32_t *dev_template = 0;
		uint32_t *dev_list = 0;

		unsigned int image_bytes = image_size * 4;
		unsigned int colors_bytes = colors_size * 4;

		hipError_t cudaStatus;

		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			printf("hipSetDevice failed!");
			goto Error;
		}

		// Allocate GPU buffers for three images (two input, one output) and the unique colors array.
		cudaStatus = hipMalloc((void**)&dev_out, image_bytes);
		if (cudaStatus != hipSuccess) {
			printf("devOut hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_colors, image_bytes);
		if (cudaStatus != hipSuccess) {
			printf("devColors hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_template, image_bytes);
		if (cudaStatus != hipSuccess) {
			printf("devTemplate hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_list, colors_bytes);
		if (cudaStatus != hipSuccess) {
			printf("devList hipMalloc failed!");
			goto Error;
		}

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy((void*)dev_colors, (void*)colors_image, image_bytes, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("devColors hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy((void*)dev_template, (void*)template_image, image_bytes, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("devTemplate hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy((void*)dev_list, (void*)colors_list, colors_bytes, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("devList hipMemcpy failed!");
			goto Error;
		}

		cudaStatus = hipMemset((void*)dev_out, 0, image_bytes);
		if (cudaStatus != hipSuccess) {
			printf("dev_out memset failed!");
			goto Error;
		}

		// Launch a kernel on the GPU with one thread for each element.
		imageMapKernel <<<BLOCKS, THREADS>>> (
			dev_out,
			dev_colors,
			dev_template,
			dev_list,
			image_size,
			colors_size);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			printf("imageMapKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// cudaDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			printf("hipDeviceSynchronize returned error code %d after launching imageMapKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(out_image, dev_out, image_bytes, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			printf("outImage hipMemcpy failed!");
			goto Error;
		}

	Error:
		hipFree(dev_out);
		hipFree(dev_colors);
		hipFree(dev_template);
		hipFree(dev_list);

		return cudaStatus;
	}


	__declspec(dllexport)
	int serialImageTemplateMap(uint8_t* out_image,
		uint8_t* colors_image,
		uint8_t* template_image,
		uint8_t* colors_list,
		unsigned int image_size,
		unsigned int colors_size) {
		
		uint32_t* temp = (uint32_t*)template_image;
		uint32_t* list = (uint32_t*)colors_list;

		uint32_t compare_color = 0;
		float average_r = 0;
		float average_g = 0;
		float average_b = 0;

		unsigned int color_index = 0;
		for (int i = 0; i < colors_size; i++) {
			compare_color = list[i];
			int x = 0;
			// Search for the first color
			for (x = 0; x < image_size; x++) {
				if (temp[x] == compare_color) {
					average_r = (float)colors_image[x * 4 + 0]; // Future improvment: read from the array once and do the binary arithmetic.
					average_g = (float)colors_image[x * 4 + 1];
					average_b = (float)colors_image[x * 4 + 2];
					break;
				}
			}

			// Average in the rest of the colors
			for (; x < image_size; x++) {
				if (temp[x] == compare_color) {
					average_r = (average_r + (float)colors_image[x * 4 + 0])/2.0f;
					average_g = (average_g + (float)colors_image[x * 4 + 1])/2.0f;
					average_b = (average_b + (float)colors_image[x * 4 + 2])/2.0f;
				}
			}

			// Write out the final average color
			for (x = 0; x < image_size; x++) {
				if (temp[x] == compare_color) {
					out_image[x * 4 + 0] = (uint8_t)average_r;
					out_image[x * 4 + 1] = (uint8_t)average_g;
					out_image[x * 4 + 2] = (uint8_t)average_b;
					out_image[x * 4 + 3] = 255;
				}
			}
		}
		return 0;
	}

}
